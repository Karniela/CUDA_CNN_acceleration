#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_complex.h>

#define M_PI 3.14159265358979323846

__global__ void dftKernel(float* input_real, float* input_imag, float* output_real, float* output_imag, int N) {
    int k = threadIdx.x + blockIdx.x * blockDim.x; // Global thread ID

    if (k < N) {
        float sum_real = 0.0f;
        float sum_imag = 0.0f;

        for (int n = 0; n < N; ++n) {
            float angle = -2.0f * M_PI * k * n / N;
            float cos_val = cosf(angle);
            float sin_val = sinf(angle);

            sum_real += input_real[n] * cos_val - input_imag[n] * sin_val;
            sum_imag += input_real[n] * sin_val + input_imag[n] * cos_val;
        }

        // Store results in output arrays
        output_real[k] = sum_real;
        output_imag[k] = sum_imag;
    }
}


void computeDFT(float* h_input_real, float* h_input_imag, float* h_output_real, float* h_output_imag, int N) {
    // Allocate device memory
    float* d_input_real, * d_input_imag, * d_output_real, * d_output_imag;
    hipMalloc(&d_input_real, N * sizeof(float));
    hipMalloc(&d_input_imag, N * sizeof(float));
    hipMalloc(&d_output_real, N * sizeof(float));
    hipMalloc(&d_output_imag, N * sizeof(float));

    // Copy input data to device
    hipMemcpy(d_input_real, h_input_real, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_input_imag, h_input_imag, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    dftKernel << <numBlocks, blockSize >> > (d_input_real, d_input_imag, d_output_real, d_output_imag, N);

    // Copy results back to host
    hipMemcpy(h_output_real, d_output_real, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_imag, d_output_imag, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input_real);
    hipFree(d_input_imag);
    hipFree(d_output_real);
    hipFree(d_output_imag);
}



int main() {
    // Example: DFT of a small array
    const int N = 1024;
    float h_input_real[N];
    float h_input_imag[N];
    for (int i = 0; i < N; i++) {
        h_input_real[i] = i + 1;
        h_input_imag[i] = 0;
    }

    float h_output_real[N];
    float h_output_imag[N];

    // Compute DFT
    computeDFT(h_input_real, h_input_imag, h_output_real, h_output_imag, N);


    // Display results
    std::cout << "DFT Results:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << "X[" << i << "] = " << h_output_real[i] << " + " << h_output_imag[i] << "i\n";
    }

    return 0;
}
